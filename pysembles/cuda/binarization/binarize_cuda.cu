#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include <cstdint>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>

#define DEBUG_1D 0
#define DEBUG_THREAD_INFO_FLOAT32 0
#define DEBUG_THREAD_INFO_INT32 0
#define DEBUG_BITS 0
#define DEBUG_SEEDS 0

template <typename scalar_t>
__global__ void binarize_kernel(
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> input) {

  // handle access indices
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int d = blockIdx.y * blockDim.y + threadIdx.y;
  const int e = blockIdx.z * blockDim.z + threadIdx.z;

  // Python version
  //output[input > 0] = 1
  //output[input <= 0] = -1
    if ((c < input.size(0)) && (d < input.size(1)) && (e < input.size(2))) {
        if (input[c][d][e] > 0) {
            input[c][d][e] = 1;
        } else {
            input[c][d][e] = -1;
        }
    }
}

std::vector<torch::Tensor> binarize_cuda(torch::Tensor input) {
  // The number of thread blocks in a grid is usually dictated by the size of the data being processed, which typically exceeds the number of processors in the system.
  // dim3 threadsPerBlock(8,8,8)
  // <<<number of blocks per grid, number of threads ber block>>>
  // grid is created with enough blocks to have one thread per matrix element

  // https://devtalk.nvidia.com/default/topic/1028226/how-many-concurrent-threads-are-running-on-my-geforce-gtx-1080-ti-/
  const int input_size_x = input.size(0);
  const int input_size_y = input.size(1);
  const int input_size_z = input.size(2);
  int threads_x = 8; // per block, 8
  int threads_y = 8; // per block, 8
  int threads_z = 8; // per block, 8

  #if DEBUG_1D
    threads_x = 1;
    threads_y = 1;
    threads_z = 1;
  #endif

  const dim3 threads(threads_x,threads_y, threads_z);
  const dim3 blocks((input_size_x + threads_x - 1) / threads_x,
                    (input_size_y + threads_y - 1) / threads_y,
                    (input_size_z + threads_z - 1) / threads_z);

  AT_DISPATCH_ALL_TYPES(input.type(), "binarize_cuda", ([&] {
    binarize_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>()
    );
  }));

  return {input};
}